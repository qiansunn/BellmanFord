#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <iomanip>
#include <cstring>
#include <sys/time.h>

#include <hip/hip_runtime.h>


using std::string;
using std::cout;
using std::endl;

#define INF 1000000

/*
 * This is a CHECK function to check CUDA calls
 */
#define CHECK(call)                                                            \
    {                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
                exit(1);                                                               \
    }                                                                          \
    }

/**
 * utils is a namespace for utility functions
 * including I/O (read input file and print results) and matrix dimension convert(2D->1D) function
 */
namespace utils {
int N; //number of vertices
int *mat; // the adjacency matrix

void abort_with_error_message(string msg) {
    std::cerr << msg << endl;
    abort();
}

//translate 2-dimension coordinate to 1-dimension
int convert_dimension_2D_1D(int x, int y, int n) {
    return x * n + y;
}

int read_file(string filename) {
    std::ifstream inputf(filename, std::ifstream::in);
    if (!inputf.good()) {
        abort_with_error_message("ERROR OCCURRED WHILE READING INPUT FILE");
    }
    inputf >> N;
    //input matrix should be smaller than 20MB * 20MB (400MB, we don't have too much memory for multi-processors)
    assert(N < (1024 * 1024 * 20));
    mat = (int *) malloc(N * N * sizeof(int));
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) {
            inputf >> mat[convert_dimension_2D_1D(i, j, N)];
        }
    return 0;
}

int print_result(bool has_negative_cycle, int *dist) {
    std::ofstream outputf("output.txt", std::ofstream::out);
    if (!has_negative_cycle) {
        for (int i = 0; i < N; i++) {
            if (dist[i] > INF)
                dist[i] = INF;
            outputf << dist[i] << '\n';
        }
        outputf.flush();
    } else {
        outputf << "FOUND NEGATIVE CYCLE!" << endl;
    }
    outputf.close();
    return 0;
}
}//namespace utils

__global__ void bellman_ford_one_iter(int n, int *d_mat, int *d_dist, bool *d_has_next, int iter_num){
    int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
    int elementSkip = blockDim.x * gridDim.x;

    if(global_tid >= n) return;
    for(int u = 0 ; u < n ; u ++){
        for(int v = global_tid; v < n; v+= elementSkip){
            int weight = d_mat[u * n + v];
            if(weight < INF){
                int new_dist = d_dist[u] + weight;
                if(new_dist < d_dist[v]){
                    d_dist[v] = new_dist;
                    *d_has_next = true;
                }
            }
        }
    }
}

/**
 * Bellman-Ford algorithm. Find the shortest path from vertex 0 to other vertices.
 * @param blockPerGrid number of blocks per grid
 * @param threadsPerBlock number of threads per block
 * @param n input size
 * @param *mat input adjacency matrix
 * @param *dist distance array
 * @param *has_negative_cycle a bool variable to recode if there are negative cycles
 */
void bellman_ford(int blocksPerGrid, int threadsPerBlock, int n, int *mat, int *dist, bool *has_negative_cycle) {
    dim3 blocks(blocksPerGrid);
    dim3 threads(threadsPerBlock);

    int iter_num = 0;
    int *d_mat, *d_dist;
    bool *d_has_next, h_has_next;

    hipMalloc(&d_mat, sizeof(int) * n * n);
    hipMalloc(&d_dist, sizeof(int) *n);
    hipMalloc(&d_has_next, sizeof(bool));

    *has_negative_cycle = false;

    for(int i = 0 ; i < n; i ++){
        dist[i] = INF;
    }

    dist[0] = 0;
    hipMemcpy(d_mat, mat, sizeof(int) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(d_dist, dist, sizeof(int) * n, hipMemcpyHostToDevice);

    for(;;){
        h_has_next = false;
        hipMemcpy(d_has_next, &h_has_next, sizeof(bool), hipMemcpyHostToDevice);

        bellman_ford_one_iter<<<blocks, threads>>>(n, d_mat, d_dist, d_has_next, iter_num);
        CHECK(hipDeviceSynchronize());
        hipMemcpy(&h_has_next, d_has_next, sizeof(bool), hipMemcpyDeviceToHost);

        iter_num++;
        if(iter_num >= n-1){
            *has_negative_cycle = true;
            break;
        }
        if(!h_has_next){
            break;
        }

    }
    if(! *has_negative_cycle){
        hipMemcpy(dist, d_dist, sizeof(int) * n, hipMemcpyDeviceToHost);
    }

    hipFree(d_mat);
    hipFree(d_dist);
    hipFree(d_has_next);
}

// CPU-based Bellman-Ford algorithm
void bellman_ford_cpu(int n, int *mat, int *dist, bool *has_negative_cycle) {
    *has_negative_cycle = false;

    for (int i = 0; i < n; ++i) {
        dist[i] = INF;
    }
    dist[0] = 0;

    for (int i = 0; i < n - 1; ++i) {
        for (int u = 0; u < n; ++u) {
            for (int v = 0; v < n; ++v) {
                int weight = mat[u * n + v];
                if (weight < INF) {
                    int new_dist = dist[u] + weight;
                    if (new_dist < dist[v]) {
                        dist[v] = new_dist;
                    }
                }
            }
        }
    }

    // Check for negative cycles
    for (int u = 0; u < n; ++u) {
        for (int v = 0; v < n; ++v) {
            int weight = mat[u * n + v];
            if (weight < INF) {
                int new_dist = dist[u] + weight;
                if (new_dist < dist[v]) {
                    *has_negative_cycle = true;
                    return;
                }
            }
        }
    }
}

// Function to identify GPU
string identify_gpu() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        return "No GPU found";
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); // Assuming only one GPU is used
    return prop.name;
}

int main(int argc, char **argv) {
    if (argc <= 1) {
        utils::abort_with_error_message("INPUT FILE WAS NOT FOUND!");
    }
    if (argc <= 3) {
        utils::abort_with_error_message("blocksPerGrid or threadsPerBlock WAS NOT FOUND!");
    }

    string filename = argv[1];
    int blockPerGrid = atoi(argv[2]);
    int threadsPerBlock = atoi(argv[3]);

    int *dist;
    bool has_negative_cycle = false;

    assert(utils::read_file(filename) == 0);
    dist = (int *) calloc(sizeof(int), utils::N);

    // CPU and GPU timing variables
    float gpu_ms_wall, cpu_ms_wall;

    // Start GPU timer
    timeval start_wall_time_t_gpu, end_wall_time_t_gpu;
    hipDeviceReset();
    gettimeofday(&start_wall_time_t_gpu, nullptr);
    // Bellman-Ford algorithm on GPU
    bellman_ford(blockPerGrid, threadsPerBlock, utils::N, utils::mat, dist, &has_negative_cycle);
    gettimeofday(&end_wall_time_t_gpu, nullptr);
    gpu_ms_wall = ((end_wall_time_t_gpu.tv_sec - start_wall_time_t_gpu.tv_sec) * 1000 * 1000
            + end_wall_time_t_gpu.tv_usec - start_wall_time_t_gpu.tv_usec) / 1000.0;

    // Start CPU timer
    timeval start_wall_time_t_cpu, end_wall_time_t_cpu;
    gettimeofday(&start_wall_time_t_cpu, nullptr);
    // Bellman-Ford algorithm on CPU
    bellman_ford_cpu(utils::N, utils::mat, dist, &has_negative_cycle);
    gettimeofday(&end_wall_time_t_cpu, nullptr);
    cpu_ms_wall = ((end_wall_time_t_cpu.tv_sec - start_wall_time_t_cpu.tv_sec) * 1000 * 1000
            + end_wall_time_t_cpu.tv_usec - start_wall_time_t_cpu.tv_usec) / 1000.0;

    // Print GPU and CPU times
    cout << "Used GPU: " << identify_gpu() << endl;
    cout << "GPU TIME: " << gpu_ms_wall / 1000.0 << " s" << endl;
    cout << "CPU TIME: " << cpu_ms_wall / 1000.0 << " s" << endl;

    utils::print_result(has_negative_cycle, dist);
    free(dist);
    free(utils::mat);

    return 0;
}
